#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"


namespace gpu {
	__global__ void addKernel(int *c, const int *a, const int *b)
	{
		int i = threadIdx.x;
		c[i] = a[i] + b[i];
	}

	// 使用CUDA并行添加矢量的辅助函数。
	hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
	{
		int *dev_a = 0;
		int *dev_b = 0;
		int *dev_c = 0;
		hipError_t cudaStatus;

		// 选择在哪个GPU上运行，在多GPU系统上更改。
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// 为三个向量分配GPU缓冲区(两个输入，一个输出)。
		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// 将输入向量从主机内存复制到GPU缓冲区。
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// 在GPU上启动一个内核，每个元素使用一个线程。
		addKernel << <1, size >> > (dev_c, dev_a, dev_b);

		// 检查启动内核时是否有错误
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize等待内核完成，并返回在启动过程中遇到的任何错误。
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// 从GPU缓冲区复制输出矢量到主机内存。
		cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		return cudaStatus;
	}



}